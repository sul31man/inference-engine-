// Row-wise LayerNorm CUDA kernels
#include <hip/hip_runtime.h>

// Compute LayerNorm per row: for each row r in [0, rows), normalize
// x[r, :] with mean/var over the last dimension (cols), then apply
// y[r, c] = (x[r, c] - mean) / sqrt(var + eps) * gamma[c] + beta[c]

extern "C" __global__ void layernorm_rowwise(
    const float* __restrict__ x,
    float* __restrict__ y,
    const float* __restrict__ gamma,
    const float* __restrict__ beta,
    int rows,
    int cols,
    float eps
) {
    extern __shared__ float smem[]; // size >= blockDim.x

    int row = blockIdx.x;
    if (row >= rows) return;

    int tid = threadIdx.x;
    const float* xr = x + row * cols;
    float* yr = y + row * cols;

    // 1) mean
    float sum = 0.0f;
    for (int c = tid; c < cols; c += blockDim.x) {
        sum += xr[c];
    }
    smem[tid] = sum;
    __syncthreads();
    for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }
    float mean = smem[0] / (float)cols;

    // 2) variance
    float vsum = 0.0f;
    for (int c = tid; c < cols; c += blockDim.x) {
        float d = xr[c] - mean;
        vsum += d * d;
    }
    smem[tid] = vsum;
    __syncthreads();
    for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }
    float var = smem[0] / (float)cols;
    float inv_std = rsqrtf(var + eps);

    // 3) normalize and affine
    for (int c = tid; c < cols; c += blockDim.x) {
        float n = (xr[c] - mean) * inv_std;
        float g = gamma ? gamma[c] : 1.0f;
        float b = beta ? beta[c] : 0.0f;
        yr[c] = n * g + b;
    }
}

// Optional scalar-parameter variant (broadcast gamma/beta scalars)
extern "C" __global__ void layernorm_rowwise_scalar(
    const float* __restrict__ x,
    float* __restrict__ y,
    float gamma,
    float beta,
    int rows,
    int cols,
    float eps
) {
    extern __shared__ float smem[];
    int row = blockIdx.x;
    if (row >= rows) return;
    int tid = threadIdx.x;
    const float* xr = x + row * cols;
    float* yr = y + row * cols;

    float sum = 0.0f;
    for (int c = tid; c < cols; c += blockDim.x) sum += xr[c];
    smem[tid] = sum; __syncthreads();
    for (int s = blockDim.x >> 1; s > 0; s >>= 1) { if (tid < s) smem[tid] += smem[tid + s]; __syncthreads(); }
    float mean = smem[0] / (float)cols;

    float vsum = 0.0f;
    for (int c = tid; c < cols; c += blockDim.x) { float d = xr[c] - mean; vsum += d*d; }
    smem[tid] = vsum; __syncthreads();
    for (int s = blockDim.x >> 1; s > 0; s >>= 1) { if (tid < s) smem[tid] += smem[tid + s]; __syncthreads(); }
    float var = smem[0] / (float)cols;
    float inv_std = rsqrtf(var + eps);

    for (int c = tid; c < cols; c += blockDim.x) {
        float n = (xr[c] - mean) * inv_std;
        yr[c] = n * gamma + beta;
    }
}