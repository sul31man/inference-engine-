#include "hip/hip_runtime.h"
#include <cudaruntime.h> 
#include "infer/softmax.cpp"
#include "inference_engine/linear.cpp"

//lets build the attention kernel. this will not be an implementation of Flash attention as this should be a basic bare minimum to get things running
//this will require us to project the tokens into q,k,v, mat mull between q and k, softmax it then mat mul with v. 


__global__ void attentionKernel(float* q, float* k, float* v, int rows, int mid, int cols){
 


    int tid = threadIdx; 
    int bid = blockIdx; 
    int gid = tid + bid*blockDim; 


    

}