//this should be embarrassingly parallel 


#include <hip/hip_runtime.h> 
#include <math.h> 

__global__ void geluKernel(float* x, float* y, int n ){



    int tid = threadIdx.x;
    int bid = blockIdx.x; 
    int gid = tid + bid*blockDim.x;

    if (gid < n){

        float val = x[gid];

        val = 0.5*val*(1 + tanh(sqrt(2/M_PI)*(val + 0.044715*pow(val, 3))));
        
        y[gid] = val; 

    }
}