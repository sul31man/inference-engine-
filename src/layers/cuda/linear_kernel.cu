#include "hip/hip_runtime.h"
#include <stdlib.h> 
#include <cudaruntime.h> 


__global__ void linearKernel(float* A, float* B, float* C ,int rows, int cols, int mid){


    int globalIdx.x = threadIdx.x + blockIdx.x*blockDim.x; 

    int globalIdx.y = threadIdx.y + blockIdx.y*blockDim.y;

    if(globalIdx.x < cols & globalIdx.y < rows){
       
        //lets create a simple implementation where we don't have to 

        float value = 0.0f; 

        for(int i=0; i < mid; i++){

            value += A[globalIdx.y*mid + i] * B[cols*i + globalIdx.x];


        }

        C[globalIdx.y*cols + globalIdx.x] = value; 

    }

}