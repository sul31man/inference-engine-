#include <hip/hip_runtime.h>


__device__ float warp_reduce_sum(float val) {
    // Warp-level reduction using shuffle primitives
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}


__global__ void softmaxKernel(float* logits, int n, float* softmax) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int gid = tid + blockDim.x * bid;
    int wid = tid % 32;
    int lane = tid / 32;
    int numWarps = (blockDim.x + 31) / 32;

    if (gid < n) {
        __shared__ float tile[32]; // Max warps per block is typically 32

        float val = logits[gid];

        // Find max value first for numerical stability
        float maxVal = val;
        for (int offset = 16; offset > 0; offset >>= 1) {
            maxVal = fmaxf(maxVal, __shfl_down_sync(0xffffffff, maxVal, offset));
        }
        maxVal = __shfl_sync(0xffffffff, maxVal, 0); // Broadcast to all threads in warp

        // Store warp max in shared memory
        if (wid == 0 && lane < numWarps) {
            tile[lane] = maxVal;
        }
        __syncthreads();

        // Find global max across all warps
        float globalMax = 0.0f;
        if (tid < numWarps) {
            globalMax = tile[tid];
        }
        for (int offset = 16; offset > 0; offset >>= 1) {
            if (tid < numWarps) {
                globalMax = fmaxf(globalMax, __shfl_down_sync(0xffffffff, globalMax, offset));
            }
        }
        globalMax = __shfl_sync(0xffffffff, globalMax, 0);

        // Compute exp(val - globalMax)
        val = expf(val - globalMax);

        // Compute sum of exponentials
        float warpSum = warp_reduce_sum(val);

        if (wid == 0 && lane < numWarps) {
            tile[lane] = warpSum;
        }
        __syncthreads();

        float sum = 0.0f;
        for (int i = 0; i < numWarps; i++) {
            sum += tile[i];
        }

        // Normalize
        val = val / sum;
        softmax[gid] = val;
    }
}